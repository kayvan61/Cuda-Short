#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "shortestKernels.cu"
#ifdef TIMING
#include "CPU_short.hpp"
#include "timingTests.hpp"
#endif
#ifdef FUNC_TEST
#include "CPU_short.hpp"
#include "funcTests.hpp"
#endif



int* genTestAdjMat(int*);
void runTimingTest();
void runCPUTimingTest();
int* read_input(const char* input);

int main() {

#ifdef TIMING
  runTimingTest();
  runCPUTimingTest();
  return 0;
#endif

#ifdef FUNC_TEST
  runFuncTests();
  return 0;
#endif

  int*  adjMat;
  int*  shortestOut;
  int   gSize;
  int   startingNode = 0;

  int*  _d_adjMat;
  int*  _d_outVec;
  int*  _d_unvisited;
  int*  _d_frontier;
  int*  _d_estimates;
  int*  _d_delta;
  int*  _d_minOutEdge;

  adjMat      = genTestAdjMat(&gSize);
  shortestOut = (int*)malloc(sizeof(int) * gSize);

  hipMalloc((void**) &_d_adjMat,     sizeof(int) * gSize * gSize);
  hipMalloc((void**) &_d_outVec,     sizeof(int) * gSize);
  hipMalloc((void**) &_d_unvisited,  sizeof(int) * gSize);
  hipMalloc((void**) &_d_frontier,   sizeof(int) * gSize);
  hipMalloc((void**) &_d_estimates,  sizeof(int) * gSize);
  hipMalloc((void**) &_d_minOutEdge, sizeof(int) * gSize);
  hipMalloc((void**) &_d_delta,      sizeof(int) * gSize);

  hipMemcpy((void*)_d_adjMat, (void*)adjMat, sizeof(int) * gSize * gSize, hipMemcpyHostToDevice);
  hipMemset((void*)_d_outVec,             0, sizeof(int) * gSize);
  hipMemset((void*)_d_unvisited,          0, sizeof(int) * gSize);
  hipMemset((void*)_d_frontier,           0, sizeof(int) * gSize);
  hipMemset((void*)_d_estimates,          0, sizeof(int) * gSize);
  hipMemset((void*)_d_minOutEdge,         0, sizeof(int) * gSize);

  doShortest( adjMat,
	      shortestOut,
	      gSize,
	      startingNode,
	      _d_adjMat,
	      _d_outVec,
	      _d_unvisited,
	      _d_frontier,
	      _d_estimates,
	      _d_delta,
	      _d_minOutEdge);

  hipFree(_d_adjMat);
  hipFree(_d_outVec);
  hipFree(_d_unvisited);
  hipFree(_d_frontier);
  hipFree(_d_estimates);
  hipFree(_d_minOutEdge);
  hipFree(_d_delta);

  free(adjMat);
  free(shortestOut);
}

int* genTestAdjMat(int* gSize) {
  *gSize = 5;
  int temp[49] = {0, 10, 0, 5, 0,
		  0, 0, 1, 2, 0,
		  0, 0, 0, 0, 4,
		  0, 3, 9, 0, 2,
		  7, 0, 6, 0, 0};
  for(int i = 0; i < 49; i++) {
    if(temp[i] == 0) {
      temp[i] = -1;
    }
  }

  int* ret = (int*)malloc(49 * sizeof(int));

  memcpy(ret, temp, sizeof(int) * 49);

  return ret;
}

int* read_input(const char* input) {
    int arraySize = 0;
    int num;
    int index = 0;
    FILE* inputF = fopen(input, "r");
    while (fscanf(inputF, "%d, ", &num) != EOF) {
        arraySize++;
    }

    int* in = (int*)malloc(arraySize * sizeof(int));
    rewind(inputF);
    while (fscanf(inputF, "%d, ", &in[index]) != EOF) {
        index++;
    }

    for (int i = 0; i < arraySize; i++) {
        printf("%d ", in[i]);
    }

    return in;
}
