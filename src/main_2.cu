#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include "shortestKernels.cu"
#ifdef TIMING
#include "CPU_short.hpp"
#include "timingTests.hpp"
#endif
#ifdef FUNC_TEST
#include "CPU_short.hpp"
#include "funcTests.hpp"
#endif



int* genTestAdjMat(int*);
void runTimingTest();
void runCPUTimingTest();
int* read_input(const char* input, int*, int*);
void write_output (FILE* file, const int* const outArr, int outArrSize);


int main(int argc, char **argv) {

#ifdef TIMING
  runTimingTest();
  runCPUTimingTest();
  return 0;
#endif

#ifdef FUNC_TEST
  runFuncTests();
  return 0;
#endif
#ifdef DEMO
  FILE* outputFile = fopen("output.txt", "w");
  int   gSize;
  
  int*  adjMat;
  int*  shortestOut;
  int   startingNode = 0;

  int*  _d_adjMat;
  int*  _d_outVec;
  int*  _d_unvisited;
  int*  _d_frontier;
  int*  _d_estimates;
  int*  _d_delta;
  int*  _d_minOutEdge;
  if(argc == 2){
    adjMat      = read_input(argv[1], &gSize, &startingNode);
  } else {
    adjMat      = genTestAdjMat(&gSize);
  }
  shortestOut = (int*)malloc(sizeof(int) * gSize);
  
  hipMalloc((void**) &_d_adjMat,     sizeof(int) * gSize * gSize);
  hipMalloc((void**) &_d_outVec,     sizeof(int) * gSize);
  hipMalloc((void**) &_d_unvisited,  sizeof(int) * gSize);
  hipMalloc((void**) &_d_frontier,   sizeof(int) * gSize);
  hipMalloc((void**) &_d_estimates,  sizeof(int) * gSize);
  hipMalloc((void**) &_d_minOutEdge, sizeof(int) * gSize);
  hipMalloc((void**) &_d_delta,      sizeof(int) * gSize);

  hipMemcpy((void*)_d_adjMat, (void*)adjMat, sizeof(int) * gSize * gSize, hipMemcpyHostToDevice);
  hipMemset((void*)_d_outVec,             0, sizeof(int) * gSize);
  hipMemset((void*)_d_unvisited,          0, sizeof(int) * gSize);
  hipMemset((void*)_d_frontier,           0, sizeof(int) * gSize);
  hipMemset((void*)_d_estimates,          0, sizeof(int) * gSize);
  hipMemset((void*)_d_minOutEdge,         0, sizeof(int) * gSize);

  doShortest( adjMat,
	      shortestOut,
	      gSize,
	      startingNode,
	      _d_adjMat,
	      _d_outVec,
	      _d_unvisited,
	      _d_frontier,
	      _d_estimates,
	      _d_delta,
	      _d_minOutEdge);

  write_output(outputFile, shortestout, gSize);
  hipFree(_d_adjMat);
  hipFree(_d_outVec);
  hipFree(_d_unvisited);
  hipFree(_d_frontier);
  hipFree(_d_estimates);
  hipFree(_d_minOutEdge);
  hipFree(_d_delta);

  free(adjMat);
  free(shortestOut);
  return 0;
#endif
}

int* genTestAdjMat(int* gSize) {
  *gSize = 5;
  int temp[49] = {0, 10, 0, 5, 0,
		  0, 0, 1, 2, 0,
		  0, 0, 0, 0, 4,
		  0, 3, 9, 0, 2,
		  7, 0, 6, 0, 0};
  for(int i = 0; i < 49; i++) {
    if(temp[i] == 0) {
      temp[i] = -1;
    }
  }

  int* ret = (int*)malloc(49 * sizeof(int));

  memcpy(ret, temp, sizeof(int) * 49);

  return ret;
}

int* read_input(const char* input, int* gSize, int* srcNode) {
    int index = 0;
    FILE* inputF = fopen(input, "r");
    
    fscanf(inputF, "%d", gSize);
    fscanf(inputF, "%d", srcNode);
    
    int* in = (int*)malloc((*gSize) * (*gSize) * sizeof(int));
    
    while (fscanf(inputF, "%d, ", &in[index]) != EOF) {
        index++;
    }

#ifndef NO_PRINT
    for(int i = 0; i < *gSize; i++) {
      for(int j = 0; j < *gSize; j++) {
	printf("%d ", in[i* (*gSize) + j]);
      }
      printf("\n");
    }
#endif
    
    return in;
}

void write_output (FILE* file, const int* const outArr, int outArrSize) {
	for(int i = 0; i < outArrSize - 1; i++) {
		fprintf(file, "%d, ", outArr[i]);
	}
	fprintf(file, "%d", outArr[outArrSize-1]);
}